#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

namespace cudascii {

    __global__ void pixel_to_ascii(int *r, int *g, int *b) {

        // Thread index
        int i = threadIdx.x + blockIdx.x * blockDim.x;

        float c_linear, c_srgb;
        int gray_index;
        char out;
        
        // Standard linear combination
        c_linear = RED_WEIGHT*(r/255.) + GREEN_WEIGHT*(g/255.) + BLUE_WEIGHT*(b/255.);
        
        // If gray level is very dark, use linear scaling
        if (c_linear <= CONVERSION_THRESHOLD)
            c_srgb = BELOW_THRESHOLD_SCALAR * c_linear;

        // Non linear scaling to adjust for gamma exposure
        else:
            c_srgb = ABOVE_THRESHOLD_SCALAR * powf(c_linear,ABOVE_THRESHOLD_EXPONENT) + ABOVE_THRESHOLD_OFFSET;
        
        // Scale c_srgb to the gray levels while handling an edge case of c_srgb = 1
        gray_index = (int) fmin(c_srgb * gray_levels, gray_levels - 1.);

        // Final character representing the gray level of the RGB pixel
        out = gray_level_lookup[gray_index];

    }

}
