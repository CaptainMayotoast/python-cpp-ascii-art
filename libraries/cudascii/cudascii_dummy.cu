#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cmath>
#include <string>

namespace {
// Algorithm Parameterization
// const std::string gray_levels_fine =
// "$@B%8&WM#*oahkbdpqwmZO0QLCJUYXzcvunxrjft/\\|()1{}[]?-_+~i!lI;:,\"^`. ";
const constexpr char* gray_level_lookup{"@%#*+=-:. "};
const constexpr int gray_levels = 10;
const constexpr float RED_WEIGHT = 0.2126;
const constexpr float GREEN_WEIGHT = 0.7152;
const constexpr float BLUE_WEIGHT = 0.0722;
const constexpr float CONVERSION_THRESHOLD = 0.0031308;
const constexpr float BELOW_THRESHOLD_SCALAR = 12.92;
const constexpr float ABOVE_THRESHOLD_SCALAR = 1.055;
const constexpr float ABOVE_THRESHOLD_EXPONENT = 1 / 2.4;
const constexpr float ABOVE_THRESHOLD_OFFSET = -0.055;
}  // namespace

namespace cudascii {

    __global__ void pixel_to_ascii(unsigned char *r, unsigned char *g, unsigned char *b) {

        // Thread index
        int i = threadIdx.x + blockIdx.x * blockDim.x;

        float c_linear, c_srgb;
        int gray_index;
        unsigned char out;
        
        // Standard linear combination
        c_linear = RED_WEIGHT*(r/255.) + GREEN_WEIGHT*(g/255.) + BLUE_WEIGHT*(b/255.);
        
        // If gray level is very dark, use linear scaling
        if (c_linear <= CONVERSION_THRESHOLD)
            c_srgb = BELOW_THRESHOLD_SCALAR * c_linear;

        // Non linear scaling to adjust for gamma exposure
        else
            c_srgb = ABOVE_THRESHOLD_SCALAR * powf(c_linear,ABOVE_THRESHOLD_EXPONENT) + ABOVE_THRESHOLD_OFFSET;
        
        // Scale c_srgb to the gray levels while handling an edge case of c_srgb = 1
        gray_index = (int) fmin(c_srgb * gray_levels, gray_levels - 1.);

        // Final character representing the gray level of the RGB pixel
        out = gray_level_lookup[gray_index];

    }
}